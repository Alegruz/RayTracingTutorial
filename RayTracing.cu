#include "hip/hip_runtime.h"
#include "RayTracing.cuh"

////////////////////////////////////////////////////////////////////////////////
// GPU-specific defines
////////////////////////////////////////////////////////////////////////////////
//Maps to a single instruction on G8x / G9x / G10x
#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )

//Round a / b to nearest higher integer value
int iDivUp(int a, int b);

__global__
void cudaInitCurand(hiprandState* state, unsigned long seed);

__global__
void cudaGetRandom(hiprandState* state, float* outRandom);

__global__
void cudaRenderRowsDevice(
	uchar* outBackBuffer, 
	size_t width, 
	size_t height, 
	size_t numChannels, 
	size_t samplesPerPixel,
	CudaSphere* spheres,
	size_t numSpheres
);

__device__
void cudaAt(const CudaRay& ray, float scalar, float3& outAt);

__device__
float3 cudaGetRandomFloat3();

__device__
float3 cudaGetRandomFloat3(float min, float max);

__device__
float3 cudaGetRandomUnitSphereFloat3();

__device__
float3 cudaGetRandomUnitFloat3();

__device__
void cudaGetRay(float u, float v, CudaRay& outRay);

__device__
bool cudaHasHit(CudaSphere* spheres, size_t numSpheres, const CudaRay& ray, float minScalar, float maxScalar, CudaHitRecord& outRecord);

__device__
bool cudaHasHitSphere(const CudaSphere& sphere, const CudaRay& ray, float minScalar, float maxScalar, CudaHitRecord& outRecord);

__device__
void cudaSetFaceNormal(CudaHitRecord& hitRecord, const CudaRay& ray, const float3& outwardNormal);

__device__
void cudaRayColor(CudaSphere* spheres, size_t numSpheres, CudaRay& ray, uint depth, float3& outColor);

__device__
void cudaRayColor(CudaSphere* spheres, size_t numSpheres, CudaRay& ray, float3& outColor);

//__device__
//bool cudaScatter(const CudaLambertian& lambertian, const CudaRay& ray, const CudaHitRecord& hitRecord);

__device__
void cudaWriteColor(uint8_t* pixels, const float3& pixelColor, size_t samplesPerPixel);

#define ASPECT_RATIO (16.0f / 9.0f)
#define VIEWPORT_HEIGHT (2.0f)
#define VIEWPORT_WIDTH (ASPECT_RATIO * VIEWPORT_HEIGHT)
#define FOCAL_LENGTH (1.0f)
#define MAX_BOUNCE_DEPTH (50u)

__constant__ CudaCamera camera;
__constant__ hiprandState devState[3];

/***********************/
/* CUDA ERROR CHECKING */
/***********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
		{
			exit(code);
		}
	}
}

//Round a / b to nearest higher integer value
inline int iDivUp(int a, int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

void CudaInitialize(CudaCamera* cudaCamera)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(camera), cudaCamera, sizeof(CudaCamera)));

	//cudaInitCurand<<<1, 1>>>(&devState, 1);
	//gpuErrchk(hipPeekAtLastError());
	//gpuErrchk(hipDeviceSynchronize());
}

void CudaRender(uchar* outBackBuffer, size_t width, size_t height, size_t numChannels, size_t samplesPerPixel, CudaSphere* spheres, size_t numSpheres)
{
	dim3 threads(16, 12);
	dim3 blocks(iDivUp(static_cast<int>(width), static_cast<int>(threads.x)), iDivUp(static_cast<int>(height), static_cast<int>(threads.y)));
	printf("Cuda Render\n");
	// execute the kernel
	cudaRenderRowsDevice<<<blocks, threads>>>(
		outBackBuffer,
		width,
		height,
		numChannels,
		samplesPerPixel,
		spheres,
		numSpheres
		);
}

__global__
void cudaInitCurand(hiprandState* state, unsigned long seed)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	printf("hiprand init idx: %d\n", idx);
	hiprand_init(seed, idx, 0, &state[idx]);
	printf("hiprand init\n");
}

__global__
void cudaRenderRowsDevice(
	uchar* outBackBuffer, 
	size_t width, 
	size_t height, 
	size_t numChannels, 
	size_t samplesPerPixel, 
	CudaSphere* spheres, 
	size_t numSpheres
)
{
	const int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
	const int iy = (height - 1) - IMAD(blockDim.y, blockIdx.y, threadIdx.y);

	if (ix >= width || iy >= height || ix < 0 || iy < 0)
	{
		return;
	}

	//printf("Processing: %d, %d, samples: %llu, spheres: %llu\n", ix, iy, samplesPerPixel, numSpheres);

	hiprandState rgnState;
	hiprand_init(0ull, ix * iy, 0, &rgnState);

	float3 pixelColor = make_float3(0.0f);
	CudaRay ray;
	for (uint sampleIdx = 0; sampleIdx < samplesPerPixel; ++sampleIdx)
	{
		float u = (ix + hiprand_uniform(&rgnState)) / static_cast<float>(width - 1);
		float v = (iy + hiprand_uniform(&rgnState)) / static_cast<float>(height - 1);
		cudaGetRay(u, v, ray);
		float3 color = make_float3(0.0f);
		cudaRayColor(spheres, numSpheres, ray, color);
		pixelColor += color;
	}
	cudaWriteColor(&outBackBuffer[(height - 1 - iy) * width * numChannels + ix * numChannels], pixelColor, samplesPerPixel);
}

__device__
void cudaAt(const CudaRay& ray, float scalar, float3& outAt)
{
	outAt = ray.origin + scalar * ray.direction;
}

__device__
float3 cudaGetRandomFloat3()
{
	return cudaGetRandomFloat3(0.0f, 1.0f);
}

__device__
float3 cudaGetRandomFloat3(float min, float max)
{
	static unsigned long long idx = 0;
	hiprand_init(0, idx, 0, &devState[0]);
	hiprand_init(0, idx + 1, 0, &devState[1]);
	hiprand_init(0, idx + 2, 0, &devState[2]);

	float3 result = make_float3(
		hiprand_uniform(&devState[0]) * (max - min) + min,
		hiprand_uniform(&devState[1]) * (max - min) + min,
		hiprand_uniform(&devState[2]) * (max - min) + min
	);
	idx += 3;

	//printf("result: %f, %f, %f\n", result.x, result.y, result.z);

	return result;
}

__device__
float3 cudaGetRandomUnitSphereFloat3()
{
	while (true)
	{
		float3 vec = cudaGetRandomFloat3(-1.0f, 1.0f);

		if (lengthSquared(vec) >= 1.0f)
		{
			continue;
		}

		return vec;
	}
}

__device__
float3 cudaGetRandomUnitFloat3()
{
	float3 vec = cudaGetRandomUnitSphereFloat3();
	return normalize(vec);
}

__device__
void cudaGetRay(float u, float v, CudaRay& outRay)
{
	//outRay.origin = camera.at;

	float3 at = make_float3(0.0f, 0.0f, 0.0f);
	float3 horizontal = make_float3(VIEWPORT_WIDTH, 0.0f, 0.0f);
	float3 vertical = make_float3(0.0f, VIEWPORT_HEIGHT, 0.0f);
	float3 lowerLeftCorner = at - (horizontal / 2.0f) - (vertical / 2.0f) - make_float3(0.0f, 0.0f, FOCAL_LENGTH);

	outRay.origin = at;
	outRay.direction = lowerLeftCorner + u * horizontal + v * vertical - at;
}

__device__
bool cudaHasHit(CudaSphere* spheres, size_t numSpheres, const CudaRay& ray, float minScalar, float maxScalar, CudaHitRecord& outRecord)
{
	CudaHitRecord tempRecord;
	bool bHasHitAnything = false;
	float closestSoFar = maxScalar;

	// check spheres
	for (size_t objectIdx = 0; objectIdx < numSpheres; ++objectIdx)
	{
		if (cudaHasHitSphere(spheres[objectIdx], ray, minScalar, maxScalar, tempRecord))
		{
			bHasHitAnything = true;
			if (closestSoFar > tempRecord.scalar)
			{
				closestSoFar = tempRecord.scalar;
				outRecord = tempRecord;
			}
		}
	}

	return bHasHitAnything;
}

__device__
bool cudaHasHitSphere(const CudaSphere& sphere, const CudaRay& ray, float minScalar, float maxScalar, CudaHitRecord& outRecord)
{
	float3 centerToOrigin = ray.origin - sphere.center;
	float a = dot(ray.direction, ray.direction);
	float halfB = dot(centerToOrigin, ray.direction);
	float c = dot(centerToOrigin, centerToOrigin) - sphere.radius * sphere.radius;
	float discriminant = halfB * halfB - a * c;

	if (discriminant < 0.0f)
	{
		return false;
	}

	float discrimantSqrt = sqrt(discriminant);

	// Find the nearest root that lies in the acceptable range.
	float root = (-halfB - discrimantSqrt) / a;
	if (root < minScalar || maxScalar < root)
	{
		root = (-halfB + discrimantSqrt) / a;
		if (root < minScalar || maxScalar < root)
		{
			return false;
		}
	}

	cudaAt(ray, root, outRecord.point);
	outRecord.scalar = root;
	float3 outwardNormal = (outRecord.point - sphere.center) / sphere.radius;
	cudaSetFaceNormal(outRecord, ray, outwardNormal);

	return true;
}

__device__
void cudaSetFaceNormal(CudaHitRecord& hitRecord, const CudaRay& ray, const float3& outwardNormal)
{
	hitRecord.bIsFrontFacing = dot(ray.direction, outwardNormal) < 0.0f;
	hitRecord.normal = (2.0f * static_cast<float>(hitRecord.bIsFrontFacing) - 1.0f) * outwardNormal;
}

__device__
void cudaRayColor(CudaSphere* spheres, size_t numSpheres, CudaRay& ray, uint depth, float3& outColor)
{
	CudaHitRecord record;

	if (depth == 0u)
	{
		//printf("depth: %u\n", depth);
		outColor = make_float3(0.0f);
		return;
	}

	if (cudaHasHit(spheres, numSpheres, ray, 0.000001f, INFINITY, record))
	{
		float3 target = record.point + record.normal + cudaGetRandomUnitFloat3();
		//outColor = (record.normal + make_float3(1.0f, 1.0f, 1.0f)) * 0.5f;

		ray.origin = record.point;
		ray.direction = target - record.point;
		//outColor *= 0.5f;
		//printf("[%u] has hit color: %f, %f, %f\n", depth, outColor.x, outColor.y, outColor.z);
		cudaRayColor(spheres, numSpheres, ray, depth - 1u, outColor);
		outColor *= 0.5f;
		return;
	}

	float3 unitDirection = normalize(ray.direction);
	float scalar = 0.5f * (unitDirection.y + 1.0f);

	outColor = lerp(make_float3(1.0f, 1.0f, 1.0f), make_float3(0.5f, 0.7f, 1.0f), scalar);
}

__device__
void cudaRayColor(CudaSphere* spheres, size_t numSpheres, CudaRay& ray, float3& outColor)
{
	cudaRayColor(spheres, numSpheres, ray, MAX_BOUNCE_DEPTH, outColor);
}

__device__
void cudaWriteColor(uint8_t* pixels, const float3& pixelColor, size_t samplesPerPixel)
{
	float scale = 1.0f / static_cast<float>(samplesPerPixel);
	float r = sqrt(pixelColor.x * scale);
	float g = sqrt(pixelColor.y * scale);
	float b = sqrt(pixelColor.z * scale);

	pixels[0] = static_cast<uint8_t>(255.999999f * clamp(r, 0.0f, 0.999999f));
	pixels[1] = static_cast<uint8_t>(255.999999f * clamp(g, 0.0f, 0.999999f));
	pixels[2] = static_cast<uint8_t>(255.999999f * clamp(b, 0.0f, 0.999999f));
}